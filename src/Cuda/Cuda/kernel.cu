#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

#define SIZE	1024

__global__ void add(int *out, const int *in_a, const int *in_b)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < SIZE)
		out[idx] = in_a[idx] + in_b[idx];
}

int main()
{

	float time = 0;

	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];
	int *d_a, *d_b, *d_c;

	// Initialize input vectors
	for (size_t i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	// Allocate and copy memory on device
	size_t size = SIZE * sizeof(int*);
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


	// Timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Launch kernel
	dim3 dimGrid(1);
	dim3 dimBlock(SIZE);

	hipEventRecord(start);
	add << <dimGrid, dimBlock >> > (d_c, d_a, d_b);
	hipDeviceSynchronize();
	hipEventRecord(stop);

	// Copy back result
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);


	// Assert correct result
	for (size_t i = 0; i < SIZE; i++) {
		if (c[i] != a[i] + b[i]) {
			std::cout << "ERROR AT INDEX " << i << ": \t" << a[i] << '+' << b[i] << "!=" << c[i] << std::endl;
			delete a; delete b; delete c;
			hipFree(d_a); hipFree(d_b); hipFree(d_c);
			return EXIT_FAILURE;
		}
	}

	std::cout << "Success!" << std::endl;
	std::cout << "Done in " << time << "ms" << std::endl;

	delete a; delete b; delete c;
	hipFree(d_a); hipFree(d_b); hipFree(d_c);


	return EXIT_SUCCESS;
}