#include "hip/hip_runtime.h"
#include "CudaCode.cuh"


__global__ void add(int *out, const int *in_a, const int *in_b)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < SIZE)
		out[idx] = in_a[idx] + in_b[idx];
}

void temp::CudaHello(){
	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];
	int *d_a, *d_b, *d_c;

	// Initialize input vectors
	for (size_t i = 0; i < SIZE; i++) {
		a[i] = i;
		b[i] = 2 * i;
		c[i] = -1;
	}

	// Allocate and copy memory on device
	size_t size = SIZE * sizeof(int*);
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch kernel
	dim3 dimGrid(1);
	dim3 dimBlock(SIZE);

	add << <dimGrid, dimBlock >> > (d_c, d_a, d_b);
	hipDeviceSynchronize();
	
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	
	for (size_t i = 0; i < 10; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	delete a; delete b; delete c;
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
}