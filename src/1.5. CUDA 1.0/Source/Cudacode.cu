#include "hip/hip_runtime.h"
#include "Cudacode.cuh"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){

		Particle *p = &ParticlesContainer[i];

		p->px += p->vx * simspeed*dt;
		p->py += p->vy * simspeed*dt;
		p->pz += p->vz * simspeed*dt;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p->px;
		g_particule_position_size_data[i * 3 + 1] = p->py;
		g_particule_position_size_data[i * 3 + 2] = p->pz;
	}


}

void CUDAUpdatePositions(Particle *p_container, GLfloat *g_particule_position_size_data, unsigned int MAX_PARTICLES, float dt){

	
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation
	
	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	gpuErrchk(hipMalloc((void**)&d_ParticlesContainer, size));
	gpuErrchk(hipMemcpy(d_ParticlesContainer, p_container, size, hipMemcpyHostToDevice));

	
	
	// Vertex buffer
	gpuErrchk(hipMalloc((void**)&d_positions, buffer_size));
	gpuErrchk(hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice));

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES / 1024);
	dim3 dimBlock(1024);

	updatePositionKernel << < dimGrid, dimBlock >> >(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();

	// retrieve the results
	gpuErrchk(hipMemcpy(p_container, d_ParticlesContainer, size, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost));
	

	hipFree(d_ParticlesContainer);
	hipFree(d_positions);

}



__global__ void updateForceKernel(OctreeNode *node){
	
}

void CUDACalcForces(OctreeNode *node){
	

	OctreeNode *d_node;
	gpuErrchk(hipMalloc((void**)&d_node, sizeof(OctreeNode)));
	gpuErrchk(hipMemcpy(d_node, node, sizeof(OctreeNode), hipMemcpyHostToDevice));

	updateForceKernel <<<1, 1 >>>(d_node);
	

	hipFree(d_node);
}