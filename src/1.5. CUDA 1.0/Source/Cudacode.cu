#include "hip/hip_runtime.h"
#include "Cudacode.cuh"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		//printf("CUDAGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		printf("CUDA::ERROR %s line %d: %s\n", file, line, hipGetErrorString(code));
		if (abort) exit(code);
	}
}


__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){

		Particle *p = &ParticlesContainer[i];

		p->px += p->vx * simspeed*dt;
		p->py += p->vy * simspeed*dt;
		p->pz += p->vz * simspeed*dt;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p->px;
		g_particule_position_size_data[i * 3 + 1] = p->py;
		g_particule_position_size_data[i * 3 + 2] = p->pz;
	}


}

void CUDAUpdatePositions(Particle *p_container, GLfloat *g_particule_position_size_data, unsigned int MAX_PARTICLES, float dt){

	
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation
	
	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	gpuErrchk(hipMalloc((void**)&d_ParticlesContainer, size));
	gpuErrchk(hipMemcpy(d_ParticlesContainer, p_container, size, hipMemcpyHostToDevice));

	
	
	// Vertex buffer
	gpuErrchk(hipMalloc((void**)&d_positions, buffer_size));
	gpuErrchk(hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice));

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES / 1024);
	dim3 dimBlock(1024);

	updatePositionKernel << < dimGrid, dimBlock >> >(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();

	// retrieve the results
	gpuErrchk(hipMemcpy(p_container, d_ParticlesContainer, size, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost));
	

	hipFree(d_ParticlesContainer);
	hipFree(d_positions);

}







__global__ void updateForceKernel(OctreeNode *nodeContainer){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	
	for (int i = 0; i < 8; i++){
		int cInd = nodeContainer[0].childIndices[i];
		
		printf("%f, ", nodeContainer[cInd].min_x);
	}

	printf("\n");
	

}

void CUDACalcForces(OctreeNode nodeContainer[], int count){

	OctreeNode *d_container;




	gpuErrchk(hipMalloc((void**)&d_container, count * sizeof(OctreeNode)));
	gpuErrchk(hipMemcpy(d_container, nodeContainer, count*sizeof(OctreeNode), hipMemcpyHostToDevice)) ;

	updateForceKernel << <1, 1 >> > (d_container);

}