#include "hip/hip_runtime.h"
#include "Cudacode.cuh"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		//printf("CUDAGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		printf("CUDA::ERROR %s line %d: %s\n", file, line, hipGetErrorString(code));
		if (abort) exit(code);
	}
}


__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){

		Particle *p = &ParticlesContainer[i];

		p->px += p->vx * simspeed*dt;
		p->py += p->vy * simspeed*dt;
		p->pz += p->vz * simspeed*dt;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p->px;
		g_particule_position_size_data[i * 3 + 1] = p->py;
		g_particule_position_size_data[i * 3 + 2] = p->pz;
	}


}

void CUDAUpdatePositions(Particle *p_container, GLfloat *g_particule_position_size_data, unsigned int MAX_PARTICLES, float dt){

	
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation
	
	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	gpuErrchk(hipMalloc((void**)&d_ParticlesContainer, size));
	gpuErrchk(hipMemcpy(d_ParticlesContainer, p_container, size, hipMemcpyHostToDevice));

	
	
	// Vertex buffer
	gpuErrchk(hipMalloc((void**)&d_positions, buffer_size));
	gpuErrchk(hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice));

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES / 1024);
	dim3 dimBlock(1024);

	updatePositionKernel << < dimGrid, dimBlock >> >(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();

	// retrieve the results
	gpuErrchk(hipMemcpy(p_container, d_ParticlesContainer, size, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost));
	

	hipFree(d_ParticlesContainer);
	hipFree(d_positions);

}



struct point
{
	float a;
	point *lChild = nullptr;
	point *rChild = nullptr;
};


__global__ void updateForceKernel(OctreeNode *nodeContainer){
	
}

__global__ void testKernel(point *p)
{
	//printf("%f\n", p[0].lChild->a);
}

void CUDACalcForces(OctreeNode nodeContainer[]){
	
	int SIZE = 3;

	point *pointArray = (point*)malloc(SIZE * sizeof(point));
	point *d_pointArray;

	
	pointArray[0].a = 0.0f;
	pointArray[1].a = 1.1f;
	pointArray[2].a = 2.2f;

	pointArray[0].lChild = &pointArray[1];
	pointArray[0].lChild = &pointArray[2];

	hipMalloc((void**)&d_pointArray, SIZE * sizeof(point));
	hipMemcpy(d_pointArray, pointArray, SIZE * sizeof(point), hipMemcpyHostToDevice);

	
	

	// launch kernel
	testKernel << <1, 1 >> >(d_pointArray);


	// deallocate memory
	free(pointArray);
	hipFree(d_pointArray);



	/*
	OctreeNode* d_nodeContainer;
	gpuErrchk(hipMalloc((void**)&d_nodeContainer, 4 * 2048 * sizeof(OctreeNode)));
	gpuErrchk(hipMemcpy(d_nodeContainer, nodeContainer, 4 * 2048 * sizeof(OctreeNode), hipMemcpyHostToDevice));
	updateForceKernel <<<1, 1 >>>(d_nodeContainer);
	
	hipFree(d_nodeContainer);
	*/


}