#include "hip/hip_runtime.h"
#include "Cudacode.cuh"

__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){

		Particle *p = &ParticlesContainer[i];

		p->px += p->vx * simspeed*dt;
		p->py += p->vy * simspeed*dt;
		p->pz += p->vz * simspeed*dt;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p->px;
		g_particule_position_size_data[i * 3 + 1] = p->py;
		g_particule_position_size_data[i * 3 + 2] = p->pz;
	}


}

void CUDAUpdatePositions(Particle *p_container, GLfloat *g_particule_position_size_data, unsigned int MAX_PARTICLES, float dt){

	
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation

	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	hipMalloc((void**)&d_ParticlesContainer, size);
	hipMemcpy(d_ParticlesContainer, p_container, size, hipMemcpyHostToDevice);
	// Vertex buffer
	hipMalloc((void**)&d_positions, buffer_size);
	hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice);

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES / 1024);
	dim3 dimBlock(1024);

	updatePositionKernel << < dimGrid, dimBlock >> >(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();

	// retrieve the results
	hipMemcpy(p_container, d_ParticlesContainer, size, hipMemcpyDeviceToHost);
	hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost);
	
}