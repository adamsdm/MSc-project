#include "hip/hip_runtime.h"
#include "Cudacode.cuh"



__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){

		Particle *p = &ParticlesContainer[i];

		p->px += p->vx * simspeed*dt;
		p->py += p->vy * simspeed*dt;
		p->pz += p->vz * simspeed*dt;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p->px;
		g_particule_position_size_data[i * 3 + 1] = p->py;
		g_particule_position_size_data[i * 3 + 2] = p->pz;
	}


}

void CUDAUpdatePositions(Particle *p_container, GLfloat *g_particule_position_size_data, unsigned int MAX_PARTICLES, float dt){

	
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation

	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	hipMalloc((void**)&d_ParticlesContainer, size);
	hipMemcpy(d_ParticlesContainer, p_container, size, hipMemcpyHostToDevice);
	// Vertex buffer
	hipMalloc((void**)&d_positions, buffer_size);
	hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice);

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES / 1024);
	dim3 dimBlock(1024);

	updatePositionKernel << < dimGrid, dimBlock >> >(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();

	// retrieve the results
	hipMemcpy(p_container, d_ParticlesContainer, size, hipMemcpyDeviceToHost);
	hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost);
	
}


__device__ struct Cell {
	float m;
	float com_x;
	float com_y;
	float com_z;
};
__global__ void updateForceKernel(OctreeNode *node){
	
	if (node->usr_val){
		Cell *c = (Cell*) node->usr_val;
	}

}

void CUDACalcForces(OctreeNode *node){
	
	OctreeNode *d_node;
	hipMalloc((void**)&d_node, sizeof(OctreeNode));
	hipMemcpy(d_node, node, sizeof(OctreeNode), hipMemcpyHostToDevice);

	void* d_usr_data;
	hipMalloc((void**) &d_usr_data, sizeof(void*));
	hipMemcpy(d_usr_data, node->usr_val, sizeof(void*), hipMemcpyHostToDevice);

	updateForceKernel << <1, 1 >> >(d_node);
}