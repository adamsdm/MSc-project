#pragma once
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

const unsigned int SIZE = 1024;

// addition kernel
__global__ void add(const int *in_a, const int *in_b, int *out)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < SIZE)
		out[idx] = in_a[idx] + in_b[idx];
}

int main(){

	// Host pointers for io data
	int *a = new int[SIZE];
	int *b = new int[SIZE];
	int *c = new int[SIZE];

	// Device pointers
	int *d_a, *d_b, *d_c;

	for (int i = 0; i < SIZE; i++){
		a[i] = i;
		b[i] = 2*i;
	}

	// Allocate memory on the device
	const unsigned int size = SIZE * sizeof(int);
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	
	// Copy the input data to the device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	dim3 dimGrid(1);
	dim3 dimBlock(SIZE);

	add <<<dimGrid, dimBlock >>> (d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; i++){
		if (a[i] + b[i] != c[i])
			return 1;
	}
	std::cout << "Sucess!" << std::endl;

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	delete[] a; delete[] b; delete[] c;

	return 0;
}