#include "hip/hip_runtime.h"
#include "ParticleSystem.cuh"



ParticleSystem::ParticleSystem(static const unsigned int _MAX_PARTICLES) {
	MAX_PARTICLES = _MAX_PARTICLES;
	// Quad vertices
	GLfloat g_vertex_buffer_data[] = {
		-0.5f, -0.5f, 0.0f,
		0.5f, -0.5f, 0.0f,
		-0.5f, 0.5f, 0.0f,
		0.5f, 0.5f, 0.0f,
	};

	g_particule_position_size_data = new GLfloat[MAX_PARTICLES * 3];
	g_particule_color_data = new GLfloat[MAX_PARTICLES * 3];

	ParticlesContainer = new Particle[_MAX_PARTICLES];
	initParticleSystem();


	glGenVertexArrays(1, &VertexArrayID);
	glBindVertexArray(VertexArrayID);

	// The VBO containing the quad/cube
	glGenBuffers(1, &billboard_vertex_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, billboard_vertex_buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(g_vertex_buffer_data), g_vertex_buffer_data, GL_STATIC_DRAW);

	// The VBO containing the positions and sizes of the particles
	glGenBuffers(1, &particles_position_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, particles_position_buffer);
	// Initialize with empty (NULL) buffer : it will be updated later, each frame.
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 3 * sizeof(GLfloat), g_particule_position_size_data, GL_STREAM_DRAW);

	// The VBO containing the colors of the particles
	glGenBuffers(1, &particles_color_buffer);
	glBindBuffer(GL_ARRAY_BUFFER, particles_color_buffer);
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 3 * sizeof(GLfloat), g_particule_color_data, GL_STREAM_DRAW);

}

void ParticleSystem::initParticleSystem(){

	float phi, r, x, y, z;
	float offset = 200;
	float rotationSpeed = 100.0f;

	// Populate initial positions and velocities
	for (int i = 0; i < MAX_PARTICLES; i++) {

		Particle p = ParticlesContainer[i];
	
		phi = (float)rand() / RAND_MAX * 2.0f * M_PI;
		r = (float)rand() / RAND_MAX * MAX_DISTANCE;

		x = r * cos(phi);
		y = 10.0f + r * sin(phi);
		z = (rand() % (2 * 40) - (float)40);
		
		
		
		// Setup particle
		p.weight = 1.0f;
		p.px = x;
		p.py = y;
		p.pz = z;

		glm::vec3 speed = glm::vec3(0.0, 0.0, 0.0);

		
		if (i % 2 == 0){
			p.px += offset;
			speed = rotationSpeed * glm::cross(glm::vec3(offset, 0.0f, 0.0f) - glm::vec3(p.px, p.py, p.pz), glm::vec3(0.0, 0.0, 1.0));
		}
		else {
			p.px -= offset;
			speed = rotationSpeed * glm::cross(glm::vec3(-offset, 0.0f, 0.0f) - glm::vec3(p.px, p.py, p.pz), glm::vec3(0.0, 0.0, 1.0));
		}
		


		p.vx = speed.x;
		p.vy = speed.y;
		p.vz = speed.z;


		ParticlesContainer[i] = p;

		// Setup position in buffer
		g_particule_position_size_data[i * 3 + 0] = p.px;
		g_particule_position_size_data[i * 3 + 1] = p.py;
		g_particule_position_size_data[i * 3 + 2] = p.pz;

		g_particule_color_data[i * 3 + 0] = 255.0f;
		g_particule_color_data[i * 3 + 1] = 255.0f;
		g_particule_color_data[i * 3 + 2] = 255.0f;
		
	}
}

ParticleSystem::~ParticleSystem(){
	delete[] g_particule_position_size_data;
	delete[] ParticlesContainer;
}

void ParticleSystem::getBounds(float &_minx, float &_maxx, float &_miny, float &_maxy, float &_minz, float &_maxz){


	float minx = 99999999.0f;
	float maxx = -99999999.0f;

	float miny = 99999999.0f;
	float maxy = -99999999.0f;

	float minz = 99999999.0f;
	float maxz = -99999999.0f;

	for (int i = 0; i < MAX_PARTICLES; i++){
		glm::vec3 pos = glm::vec3(ParticlesContainer[i].px, ParticlesContainer[i].py, ParticlesContainer[i].pz);

		minx = min(pos.x, minx);
		maxx = max(pos.x, maxx);

		miny = min(pos.y, miny);
		maxy = max(pos.y, maxy);

		minz = min(pos.z, minz);
		maxz = max(pos.z, maxz);
	}


	_minx = minx;
	_maxx = maxx;

	_miny = miny;
	_maxy = maxy;

	_minz = minz;
	_maxz = maxz;

}

void ParticleSystem::renderBounds(){
	float minx, maxx;
	float miny, maxy;
	float minz, maxz;

	getBounds(minx, maxx, miny, maxy, minz, maxz);

	// Bad way.. :/
	float vertices[] = {
		minx, miny, minz,
		maxx, miny, minz,
		maxx, maxy, minz,
		maxx, maxy, minz,
		minx, maxy, minz,
		minx, miny, minz,

		minx, miny, maxz,
		maxx, miny, maxz,
		maxx, maxy, maxz,
		maxx, maxy, maxz,
		minx, maxy, maxz,
		minx, miny, maxz,

		maxx, maxy, maxz,
		minx, maxy, minz,
		minx, miny, minz,
		minx, miny, minz,
		minx, miny, maxz,
		minx, maxy, maxz,

		maxx, maxy, maxz,
		maxx, maxy, minz,
		maxx, miny, minz,
		maxx, miny, minz,
		maxx, miny, maxz,
		maxx, maxy, maxz,

		minx, miny, minz,
		maxx, miny, minz,
		maxx, miny, maxz,
		maxx, miny, maxz,
		minx, miny, maxz,
		minx, miny, minz,

		minx, maxy, minz,
		maxx, maxy, minz,
		maxx, maxy, maxz,
		maxx, maxy, maxz,
		maxx, maxy, maxz,
		minx, maxy, minz,
	};


	unsigned int VBO, VAO;
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);

	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

	// position attribute

	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
	glEnableVertexAttribArray(0);

	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	glDrawArrays(GL_TRIANGLES, 0, 36);
}

void ParticleSystem::render(float dt){

	
	CUDAStep(0.03);

	// Update buffer data
	glBindBuffer(GL_ARRAY_BUFFER, particles_position_buffer);
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 3 * sizeof(GLfloat), NULL, GL_STREAM_DRAW); // Buffer orphaning, a common way to improve streaming perf. See above link for details.
	glBufferSubData(GL_ARRAY_BUFFER, 0, MAX_PARTICLES * 3 * sizeof(GLfloat), g_particule_position_size_data);


	glBindBuffer(GL_ARRAY_BUFFER, particles_color_buffer);
	glBufferData(GL_ARRAY_BUFFER, MAX_PARTICLES * 3 * sizeof(GLfloat), NULL, GL_STREAM_DRAW); // Buffer orphaning, a common way to improve streaming perf. See above link for details.
	glBufferSubData(GL_ARRAY_BUFFER, 0, MAX_PARTICLES * 3 * sizeof(GLfloat), g_particule_color_data);



	// Setup attributes for the particle shader
	// 1rst attribute buffer : vertices
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, billboard_vertex_buffer);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);

	// 2nd attribute buffer : positions of particles' centers
	glEnableVertexAttribArray(1);
	glBindBuffer(GL_ARRAY_BUFFER, particles_position_buffer);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 0, (void*)0);

	// 3rd attribute buffer : particles' colors
	glEnableVertexAttribArray(2);
	glBindBuffer(GL_ARRAY_BUFFER, particles_color_buffer);
	glVertexAttribPointer(2, 3, GL_FLOAT, GL_TRUE, 0, (void*)0);


	glVertexAttribDivisor(0, 0); // particles vertices : always reuse the same 4 vertices -> 0
	glVertexAttribDivisor(1, 1); // positions : one per quad (its center)                 -> 1
	glVertexAttribDivisor(2, 1); // color : one per quad                                  -> 1


	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	glDrawArraysInstanced(GL_TRIANGLE_STRIP, 0, 4, MAX_PARTICLES);

	glDisableVertexAttribArray(0);
	glDisableVertexAttribArray(1);
	glDisableVertexAttribArray(2);

}

__global__ void updateForceKernel(Particle *p, int MAX_PARTICLES)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < MAX_PARTICLES){
		float Fx = 0; float Fy = 0; float Fz = 0;

		for (int j = 0; j < MAX_PARTICLES; j++){

			float dx = p[j].px - p[i].px;
			float dy = p[j].py - p[i].py;
			float dz = p[j].pz - p[i].pz;

			float dist = sqrt(dx*dx + dy*dy + dz*dz) + 1.0f;

			float F = (9.82 * p[i].weight * p[j].weight) / (dist + SOFTENING * SOFTENING);

			Fx += F * dx / dist;
			Fy += F * dy / dist;
			Fz += F * dz / dist;
			
		}


		p[i].Fx = Fx;
		p[i].Fy = Fy;
		p[i].Fz = Fz;

		
		// Update speed
		p[i].vx += Fx;
		p[i].vy += Fy;
		p[i].vz += Fz;
		
	}
}

__global__ void updatePositionKernel(GLfloat *g_particule_position_size_data, Particle *ParticlesContainer, int MAX_PARTICLES, float dt, float simspeed){

	int i = blockIdx.x * blockDim.x + threadIdx.x;


	if (i < MAX_PARTICLES){
		
		Particle p = ParticlesContainer[i];

		p.px = p.px + p.vx * simspeed*dt;
		p.py = p.py + p.vy * simspeed*dt;
		p.pz = p.pz + p.vz * simspeed*dt;


		ParticlesContainer[i] = p;

		// Update position buffer
		g_particule_position_size_data[i * 3 + 0] = p.px;
		g_particule_position_size_data[i * 3 + 1] = p.py;
		g_particule_position_size_data[i * 3 + 2] = p.pz;
	}
	

}

void ParticleSystem::CUDAStep(float dt){
	
	 
	int size = MAX_PARTICLES * sizeof(Particle);
	float buffer_size = MAX_PARTICLES * 3 * sizeof(GLfloat);
	float simspeed = 0.01f;	// No more than ~0.1 for a stable simulation

	// Allocate memory
	Particle *d_ParticlesContainer;
	GLfloat *d_positions;

	// Particle container
	hipMalloc((void**)&d_ParticlesContainer, size);
	hipMemcpy(d_ParticlesContainer, ParticlesContainer, size, hipMemcpyHostToDevice);

	// Vertex buffer
	hipMalloc((void**)&d_positions, buffer_size);
	hipMemcpy(d_positions, g_particule_position_size_data, buffer_size, hipMemcpyHostToDevice);

	// launch kernel
	dim3 dimGrid(MAX_PARTICLES/1024);
	dim3 dimBlock(1024);

	updateForceKernel <<< dimGrid, dimBlock >>>(d_ParticlesContainer, MAX_PARTICLES);
	hipDeviceSynchronize();
	updatePositionKernel <<< dimGrid, dimBlock >>>(d_positions, d_ParticlesContainer, MAX_PARTICLES, dt, simspeed);
	hipDeviceSynchronize();
	
	// retrieve the results
	hipMemcpy(g_particule_position_size_data, d_positions, buffer_size, hipMemcpyDeviceToHost);
	hipMemcpy(ParticlesContainer, d_ParticlesContainer, size, hipMemcpyDeviceToHost);


	// TEMP
	
	float maxF = 0.0f;
	for (size_t i = 0; i < MAX_PARTICLES; i++)
	{
		Particle p = ParticlesContainer[i];
		float F = sqrt(p.Fx * p.Fx + p.Fy*p.Fy + p.Fz*p.Fz);
		
		g_particule_color_data[i * 3 + 0] = p.Fx;
		g_particule_color_data[i * 3 + 1] = p.Fy;
		g_particule_color_data[i * 3 + 2] = p.Fz;
	}
	
	hipFree(d_ParticlesContainer); hipFree(d_positions);

}


